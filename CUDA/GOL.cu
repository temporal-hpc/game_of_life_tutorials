
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
 
#define SRAND_VALUE 1985
#define BLOCK_SIZE 16
 
__global__ void ghostRows(int dim, int *grid)
{
    // We want id ∈ [1,dim]
    int id = blockDim.x * blockIdx.x + threadIdx.x + 1;
 
    if (id <= dim)
    {
        //Copy first real row to bottom ghost row
        grid[(dim+2)*(dim+1)+id] = grid[(dim+2)+id];
        //Copy last real row to top ghost row
        grid[id] = grid[(dim+2)*dim + id];
    }
}
 
__global__ void ghostCols(int dim, int *grid)
{
    // We want id ∈ [0,dim+1]
    int id = blockDim.x * blockIdx.x + threadIdx.x;
 
    if (id <= dim+1)
    {
        //Copy first real column to right most ghost column
        grid[id*(dim+2)+dim+1] = grid[id*(dim+2)+1];
        //Copy last real column to left most ghost column 
        grid[id*(dim+2)] = grid[id*(dim+2) + dim];    
    }
}
 
__global__ void GOL(int dim, int *grid, int *newGrid)
{
    // We want id ∈ [1,dim]
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int id = iy * (dim+2) + ix;
 
    int numNeighbors;
 
    if (iy <= dim && ix <= dim) {
 
        // Get the number of neighbors for a given grid point
        numNeighbors = grid[id+(dim+2)] + grid[id-(dim+2)] //upper lower
                     + grid[id+1] + grid[id-1]             //right left
                     + grid[id+(dim+3)] + grid[id-(dim+3)] //diagonals
                     + grid[id-(dim+1)] + grid[id+(dim+1)];
 
        int cell = grid[id];
        // Here we have explicitly all of the game rules
        if (cell == 1 && numNeighbors < 2)
            newGrid[id] = 0;
        else if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3))
            newGrid[id] = 1;
        else if (cell == 1 && numNeighbors > 3)
            newGrid[id] = 0;
        else if (cell == 0 && numNeighbors == 3)
            newGrid[id] = 1;
        else
            newGrid[id] = cell;
    }
}
 
int main(int argc, char* argv[]){
    if(argc != 3){
        fprintf(stderr, "run as ./prog <n> <iter>\n\n");
        exit(EXIT_FAILURE);
    }
    int i,j,iter;
    int* h_grid; //Grid on host
    int* d_grid; //Grid on device
    int* d_newGrid; //Second grid used on device only
    int* d_tmpGrid; //tmp grid pointer used to switch between grid and newGrid
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
 
    int dim = atoi(argv[1]); //Linear dimension of our grid - not counting ghost cells
    int maxIter = atoi(argv[2]); //Number of game steps
    printf("Using n=%i  and   iter=%i\n", dim, maxIter); fflush(stdout);
 
    size_t bytes = sizeof(int)*(dim+2)*(dim+2);//2 added for periodic boundary condition ghost cells
    // Allocate host Grid used for initial setup and read back from device
    h_grid = (int*)malloc(bytes);
 
    // Allocate device grids
    hipMalloc(&d_grid, bytes);
    hipMalloc(&d_newGrid, bytes);
 
    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 1; i<=dim; i++) {
        for(j = 1; j<=dim; j++) {
            h_grid[i*(dim+2)+j] = rand() % 2;
        }
    }
 
    // Copy over initial game grid (Dim-1 threads)
    hipMemcpy(d_grid, h_grid, bytes, hipMemcpyHostToDevice);
 
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE,1);
    int linGrid = (int)ceil(dim/(float)BLOCK_SIZE);
    dim3 gridSize(linGrid,linGrid,1);
 
    dim3 cpyBlockSize(BLOCK_SIZE,1,1);
    dim3 cpyGridRowsGridSize((int)ceil(dim/(float)cpyBlockSize.x),1,1);
    dim3 cpyGridColsGridSize((int)ceil((dim+2)/(float)cpyBlockSize.x),1,1);
 
    // Main game loop
    printf("GOL GLOBAL MEM\n"); fflush(stdout);
    printf("Simulating %i iterations.......", maxIter); fflush(stdout);
    hipEventRecord(start);
    for (iter = 0; iter<maxIter; iter++) {
 
        ghostRows<<<cpyGridRowsGridSize, cpyBlockSize>>>(dim, d_grid);
        hipDeviceSynchronize();
        ghostCols<<<cpyGridColsGridSize, cpyBlockSize>>>(dim, d_grid);
        hipDeviceSynchronize();
        GOL<<<gridSize, blockSize>>>(dim, d_grid, d_newGrid);
        hipDeviceSynchronize();
 
        // Swap our grids and iterate again
        d_tmpGrid = d_grid;
        d_grid = d_newGrid;
        d_newGrid = d_tmpGrid;
    }//iter loop
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0; 
    hipEventElapsedTime(&ms, start, stop);
    printf("done: %f secs\n", ms/1000.0); fflush(stdout);
 
    // Copy back results and sum
    hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);
 
    // Sum up alive cells and print results
    int total = 0;
    for (i = 1; i<=dim; i++) {
        for (j = 1; j<=dim; j++) {
            total += h_grid[i*(dim+2)+j];
        }
    }
    printf("Total Alive: %d\n", total);
 
    // Release memory
    hipFree(d_grid);
    hipFree(d_newGrid);
    free(h_grid);
 
    return 0;
}
